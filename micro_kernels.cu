#include "hip/hip_runtime.h"
/**************************************************************************
*  GPU_microbench -- GPU reliability microbenchmarks                      *
*                                                                         *
*  Copyright 2023-24 Jose M. Badia <badia@uji.es> and                     *
*                    German Leon <leon@uji.es>                            *
*                                                                         *
*  micro_kernels.cu is part of GPU_microbench                             *
*                                                                         *
*  GPU_microbench is free software: you can redistribute it and/or modify *
*  it under the terms of the GNU General Public License as published by   *
*  the Free Software Foundation; either version 3 of the License, or      *
*  (at your option) any later version.                                    *
*                                                                         *
*  GPU_microbench is distributed in the hope that it will be useful, but  *
*  WITHOUT ANY WARRANTY; without even the implied warranty of             *
*  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU      *
*  General Public License for more details.                               *
*                                                                         *
*  You should have received a copy of the GNU General Public License      *
*  along with this program.  If not, see <http://www.gnu.org/licenses/>   *
*                                                                         *
***************************************************************************/

// System includes
#include <unistd.h>
#include <stdio.h>
#include <stdbool.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#include "micro.h"

/**
 * Micro Kernel that performs the computation using only registers.
 * Version with a given number of iterations
 */
__global__ void microKernel_reg_iter (unsigned int nit, char *vadd) {

    btype regin, regout, local;
    btype id = (blockIdx.x*blockDim.x + threadIdx.x+1);

    regin = id;
    local = id;
#pragma unroll 2 
    for (int op = 0; op < nit; ++op) {
      regout = regin*local + id;
      local = (regout-local)/regin;
    }
    vadd[(int) id - 1] = (local == id);

}

/**
 * Micro Kernel that performs the computation using only registers 
 * Version with a given time
 */
__global__ void microKernel_reg_time (unsigned int cycles, char *vadd) {
    unsigned int fin,ahora;
    btype regin, regout, local;
    btype id = (blockIdx.x*blockDim.x + threadIdx.x+1);

    ahora=myclock();
    regin = id;
    local = id;
    fin=ahora+cycles;

    while (ahora < fin  )  {   
     ahora=myclock();

     #pragma unroll 2
     for (unsigned int op=0; op< QUATUMINTERACIONES;++op){
      regout = regin*local + id;
      
      local = (regout-local)/regin; 
      }  
    }  // end while

    vadd[(int) id - 1] = (local == id);

}

/**
 * Micro Kernel that performs the computation using global memory (and cache)
 * Version with a given number of iterations 
 */
__global__ void microKernel_global_iter(int nit, char *vadd, volatile btype *global) {
    btype regin, regout;
    btype id = (blockIdx.x*blockDim.x + threadIdx.x+1);
    int idInt = SIZEROW*(int) id;

    regin = id;
    global[idInt] = id;

    #pragma unroll 2 
    for (int op = 0; op < nit; ++op) {
      regout = regin*global[idInt] + id;
      global[idInt] = (regout-global[idInt])/regin;
    }
    vadd[(int) id - 1] = ( global[idInt] == id );
}

/**
 * Micro Kernel that performs the computation using global memory (and cache)
 * Version with a given time
 */
__global__ void microKernel_global_time(unsigned int cycles, char *vadd, volatile btype *global) {
    unsigned  int fin,ahora;
    btype regin, regout;
    btype id = (blockIdx.x*blockDim.x + threadIdx.x+1);
    volatile int idInt = SIZEROW*(int) id;

    ahora=myclock();
    regin = id;
    fin=ahora+cycles;
    global[idInt] = id;
    while (ahora < fin  )  {   
      ahora=myclock();

      #pragma unroll 2 
      for (unsigned  int op = 0; op < QUATUMINTERACIONES; ++op) {
        regout = regin*global[idInt] + id;
        global[idInt] = (regout-global[idInt])/regin;
      }
    }
    vadd[(int) id - 1] = ( global[idInt] == id );
}

/**
 * Micro Kernel that performs the computation using shared memory
 * Version with a given number of iterations
 */
__global__ void microKernel_shared_iter(unsigned int nit, char *vadd) {
    
  
    btype regin, regout;
    volatile btype id = (btype) (blockIdx.x*blockDim.x + threadIdx.x + 1);

    volatile extern __shared__ btype sh[];

    regin = id;
    sh[threadIdx.x] = id;

    #pragma unroll 2 
    for (unsigned int op = 0; op < nit; ++op) {
      regout = regin*sh[threadIdx.x] + id;
      sh[threadIdx.x] = (regout-sh[threadIdx.x])/regin;
    }
    vadd[(int) id - 1 ] = (sh[threadIdx.x] == id);
}

/**
 * Micro Kernel that performs the computation using shared memory
 * Version with a given number of time
 */
__global__ void microKernel_shared_time (unsigned int cycles, char *vadd) {
    
    unsigned int fin,ahora;
    btype regin, regout;
    volatile btype id = (btype) (blockIdx.x*blockDim.x + threadIdx.x + 1);

    volatile extern __shared__ btype sh[];
    ahora=myclock();
    regin = id;
    sh[threadIdx.x] = id;
    fin=ahora+cycles;

    while (ahora < fin  )  {   
     ahora=myclock();

     #pragma unroll 2 
     for (int op = 0; op < QUATUMINTERACIONES; ++op) {
       regout = regin*sh[threadIdx.x] + id;
       sh[threadIdx.x] = (regout-sh[threadIdx.x])/regin;
     }
    } 
    vadd[(int) id - 1 ] = (sh[threadIdx.x] == id);
}


/**
  * Checks if there is any error in the result of any thread
  * cont returns the number of threads with a wrong result
  */
bool check_error(char *h_vadd, int vsize, int *cont, int *id) {
    *cont = 0;
    for (int i = 0; i < vsize; i++) 
        if (!h_vadd[i] ) {
 	   (*cont)++;
           *id = (i+1);
	}
    return (*cont == 0);
}


/**
 * Launch microKernel
 */
int launch_kernel(char *bench, int grid, int blk, unsigned int nitocycles, int time) {
    char *h_vadd;
    char *d_vadd;
    btypePtr d_global;
    int vsize = grid*blk;
   

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    h_vadd = (char *) malloc(vsize*sizeof(char));

    checkCudaErrors(hipMalloc(&d_vadd, vsize*sizeof(char)));
    checkCudaErrors(hipDeviceSynchronize());
 
    // Record the start event
    checkCudaErrors(hipEventRecord(start));

    // Execute the kernel
  
    if (!strcmp(bench, "shm") ) {
        if(time) {
            microKernel_shared_time <<<grid, blk, blk*sizeof(btype)>>>(nitocycles, d_vadd);
        }
        else {
            microKernel_shared_iter <<<grid, blk, blk*sizeof(btype)>>>(nitocycles, d_vadd);
        } 
    } else if (!strcmp(bench, "glb") ) {
        checkCudaErrors(hipMalloc(&d_global, SIZEROW*vsize*sizeof(btype)));
        if(time) {
            microKernel_global_time <<<grid, blk, blk*sizeof(btype)>>>(nitocycles, d_vadd, d_global);
        }                                                
        else {
            microKernel_global_iter <<<grid, blk, blk*sizeof(btype)>>>(nitocycles, d_vadd, d_global);
        }
    } else if (!strcmp(bench, "reg") ) {
        if(time) {
            microKernel_reg_time <<<grid, blk, blk*sizeof(btype)>>>(nitocycles, d_vadd);
        }
        else {
            microKernel_reg_iter <<<grid, blk, blk*sizeof(btype)>>>(nitocycles, d_vadd);
       } 
    } 

    // Record the stop event
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipEventRecord(stop));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    // Compute and print the performance
    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
//    printf("Kernel time= %.2f ms.\n", msecTotal);

    checkCudaErrors( hipMemcpy(h_vadd, d_vadd, vsize*sizeof(char), hipMemcpyDeviceToHost) );
    
    int cont, id;
    bool correct = check_error(h_vadd, vsize, &cont, &id);

    // Clean up memory
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipFree(d_vadd));
    if (!strcmp(bench, "glb") ) {
        checkCudaErrors(hipFree(d_global));
    }
    free(h_vadd);

    if (!correct)
      printf("FAIL Num fails: %d, Thread id: %d\n", cont, id);

    return correct;

}
